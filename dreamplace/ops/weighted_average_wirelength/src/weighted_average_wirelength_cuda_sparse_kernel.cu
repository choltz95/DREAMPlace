#include "hip/hip_runtime.h"
#include <cfloat>
#include <stdio.h>
#include "assert.h"
#include "hip/hip_runtime.h"
#include "utility/src/print.h"
#include "weighted_average_wirelength/src/functional_cuda.h"
#include "utility/src/csrmv.h"
#include "utility/src/Msg.h"

DREAMPLACE_BEGIN_NAMESPACE

template <typename T>
__global__ void multiply(const T* a, const T* b, int n, T* c)
{   
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        c[i] = a[i]*b[i];
    }
}

/// compute summation of values correlated to each pin for each net
template <typename T>
void computeNetSum(
        T** x, // length of batch x #pins
        const int* flat_netpin, // JA
        const int* netpin_start, // IA
        const T* netpin_values, // A
        int num_nets,
        int num_pins,
        int num_batch,
        T** net_sum_x // length of batch x #nets
        )
{
    // ------------------ Prepare Data for GPU sparse matrix multiplication ------------
    hipsparseStatus_t status;
    hipsparseHandle_t handle=0;
    hipsparseMatDescr_t descr=0;

    /* initialize cusparse library */
    assert( hipsparseCreate(&handle) == HIPSPARSE_STATUS_SUCCESS );
    assert( hipsparseCreateMatDescr(&descr) == HIPSPARSE_STATUS_SUCCESS );
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    const T alpha = 1.0;
    const T beta = 0.0;

    hipDeviceSynchronize();

    //hipEvent_t start, stop;
    //hipEventCreate(&start);
    //hipEventCreate(&stop);
    //hipEventRecord(start);

    for (int i = 0; i < num_batch; ++i)
    {
        /* exercise Level 2 routines (csrmv) */
        /* Multiply to get sum of pins for each net */
        status = csrmv<T>(
                handle,
                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                num_nets,
                num_pins,
                num_pins,
                &alpha,
                descr,
                netpin_values,
                netpin_start,
                flat_netpin,
                x[i],
                &beta,
                net_sum_x[i]
                );
        if (status != HIPSPARSE_STATUS_SUCCESS)
        {
            printf("[E] cusparse csrmv failed for batch %d\n", i);
            exit(-1);
        }
    }
    hipDeviceSynchronize();

    //hipEventRecord(stop);
    //hipEventSynchronize(stop);

    //float milliseconds = 0;
    //hipEventElapsedTime(&milliseconds, start, stop);
    //std::cout << "Net Sum : " << milliseconds << " milli sec" << std::endl;
}

template <typename T, typename V>
int computeWeightedAverageWirelengthCudaSparseLauncher(
        const T* x, const T* y,
        const int* flat_netpin,
        const int* netpin_start,
        const T* netpin_values,
        const int* pin2net_map,
        const unsigned char* net_mask,
        int num_nets,
        int num_pins,
        const T* inv_gamma,
        T* exp_xy, T* exp_nxy,
        T* exp_xy_sum, T* exp_nxy_sum,
        T* xyexp_xy_sum, T* xyexp_nxy_sum,
        V* xy_max, V* xy_min,
        T* partial_wl, // wirelength of each net
        const T* grad_tensor,
        T* grad_x_tensor, T* grad_y_tensor // the gradient is partial total wirelength to partial pin position
        )
{
    int thread_count = 512;
    int block_count_pins = (num_pins - 1 + thread_count) / thread_count; // separate x and y
    int block_count_nets = (num_nets - 1 + thread_count) / thread_count;

    hipError_t status;
    hipStream_t stream_nx_exp;
    hipStream_t stream_y_exp;
    hipStream_t stream_ny_exp;
    status = hipStreamCreate(&stream_y_exp);
    if (status != hipSuccess)
    {
        printf("hipStreamCreate failed for stream_y_exp\n");
        fflush(stdout);
        return 1;
    }

    if (grad_tensor)
    {
        computeWeightedAverageWirelengthGradPinByPin<<<block_count_pins, thread_count>>>(
                x, y,
                exp_xy, exp_nxy,
                exp_xy_sum, exp_nxy_sum,
                xyexp_xy_sum, xyexp_nxy_sum,
                pin2net_map,
                net_mask,
                num_nets,
                num_pins,
                inv_gamma,
                grad_tensor,
                grad_x_tensor, grad_y_tensor
                );
    }
    else
    {
        T* xyexp_xy = nullptr;
        T* xyexp_nxy = nullptr;
        status = hipMalloc((void**)&xyexp_xy, 2*num_pins*sizeof(T));
        if (status != hipSuccess)
        {
            printf("hipMalloc failed for xyexp_xy\n");
            fflush(stdout);
            return 1;
        }
        status = hipMalloc((void**)&xyexp_nxy, 2*num_pins*sizeof(T));
        if (status != hipSuccess)
        {
            printf("hipMalloc failed for xyexp_nxy\n");
            fflush(stdout);
            return 1;
        }

        status = hipStreamCreate(&stream_nx_exp);
        if (status != hipSuccess)
        {
            printf("hipStreamCreate failed for stream_nx_exp\n");
            fflush(stdout);
            return 1;
        }
        status = hipStreamCreate(&stream_ny_exp);
        if (status != hipSuccess)
        {
            printf("hipStreamCreate failed for stream_ny_exp\n");
            fflush(stdout);
            return 1;
        }

        // compute max/min
        computeMax<<<block_count_pins, thread_count>>>(
                x,
                pin2net_map,
                net_mask,
                num_pins,
                xy_max
                );
        computeMin<<<block_count_pins, thread_count, 0, stream_nx_exp>>>(
                x,
                pin2net_map,
                net_mask,
                num_pins,
                xy_min
                );
        computeMax<<<block_count_pins, thread_count, 0, stream_y_exp>>>(
                y,
                pin2net_map,
                net_mask,
                num_pins,
                xy_max+num_nets
                );
        computeMin<<<block_count_pins, thread_count, 0, stream_ny_exp>>>(
                y,
                pin2net_map,
                net_mask,
                num_pins,
                xy_min+num_nets
                );

        // compute exp and negative exp
        computeExp<<<block_count_pins, thread_count>>>(
                x,
                pin2net_map,
                net_mask,
                num_nets,
                num_pins,
                inv_gamma,
                xy_max,
                exp_xy
                );
        computeNegExp<<<block_count_pins, thread_count, 0, stream_nx_exp>>>(
                x,
                pin2net_map,
                net_mask,
                num_nets,
                num_pins,
                inv_gamma,
                xy_min,
                exp_nxy
                );
        computeExp<<<block_count_pins, thread_count, 0, stream_y_exp>>>(
                y,
                pin2net_map,
                net_mask,
                num_nets,
                num_pins,
                inv_gamma,
                xy_max+num_nets,
                exp_xy+num_pins
                );
        computeNegExp<<<block_count_pins, thread_count, 0, stream_ny_exp>>>(
                y,
                pin2net_map,
                net_mask,
                num_nets,
                num_pins,
                inv_gamma,
                xy_min+num_nets,
                exp_nxy+num_pins
                );

        // compute x*exp and x*negative exp
        multiply<<<block_count_pins, thread_count>>>(
                x,
                exp_xy,
                num_pins,
                xyexp_xy
                );
        multiply<<<block_count_pins, thread_count, 0, stream_nx_exp>>>(
                x,
                exp_nxy,
                num_pins,
                xyexp_nxy
                );
        multiply<<<block_count_pins, thread_count, 0, stream_y_exp>>>(
                y,
                exp_xy+num_pins,
                num_pins,
                xyexp_xy+num_pins
                );
        multiply<<<block_count_pins, thread_count, 0, stream_ny_exp>>>(
                y,
                exp_nxy+num_pins,
                num_pins,
                xyexp_nxy+num_pins
                );

        // compute exp sum
        // compute x exp sum
        T** pin_value_arrays = new T* [8];
        pin_value_arrays[0] = exp_xy;
        pin_value_arrays[1] = exp_xy+num_pins;
        pin_value_arrays[2] = exp_nxy;
        pin_value_arrays[3] = exp_nxy+num_pins;
        pin_value_arrays[4] = xyexp_xy;
        pin_value_arrays[5] = xyexp_xy+num_pins;
        pin_value_arrays[6] = xyexp_nxy;
        pin_value_arrays[7] = xyexp_nxy+num_pins;
        T** net_sum_x_arrays = new T* [8];
        net_sum_x_arrays[0] = exp_xy_sum;
        net_sum_x_arrays[1] = exp_xy_sum+num_nets;
        net_sum_x_arrays[2] = exp_nxy_sum;
        net_sum_x_arrays[3] = exp_nxy_sum+num_nets;
        net_sum_x_arrays[4] = xyexp_xy_sum;
        net_sum_x_arrays[5] = xyexp_xy_sum+num_nets;
        net_sum_x_arrays[6] = xyexp_nxy_sum;
        net_sum_x_arrays[7] = xyexp_nxy_sum+num_nets;
        computeNetSum(
                pin_value_arrays,
                flat_netpin,
                netpin_start,
                netpin_values,
                num_nets,
                num_pins,
                8,
                net_sum_x_arrays
                );
        delete [] pin_value_arrays;
        delete [] net_sum_x_arrays;

        // compute log sum exp
        computeXExpSumByExpSum<<<block_count_nets, thread_count>>>(
                xyexp_xy_sum,
                exp_xy_sum,
                pin2net_map,
                net_mask,
                num_nets,
                partial_wl
                );
        computeXNegExpSumByNegExpSum<<<block_count_nets, thread_count, 0, stream_nx_exp>>>(
                xyexp_nxy_sum,
                exp_nxy_sum,
                pin2net_map,
                net_mask,
                num_nets,
                partial_wl+num_nets
                );

        computeXExpSumByExpSum<<<block_count_nets, thread_count, 0, stream_y_exp>>>(
                xyexp_xy_sum+num_nets,
                exp_xy_sum+num_nets,
                pin2net_map,
                net_mask,
                num_nets,
                partial_wl+2*num_nets
                );
        computeXNegExpSumByNegExpSum<<<block_count_nets, thread_count, 0, stream_ny_exp>>>(
                xyexp_nxy_sum+num_nets,
                exp_nxy_sum+num_nets,
                pin2net_map,
                net_mask,
                num_nets,
                partial_wl+3*num_nets
                );

        // I move out the summation to use ATen
        // significant speedup is observed
        //sumArray<<<1, 1>>>(partial_wl, 2*num_nets, wl);

        status = hipStreamDestroy(stream_nx_exp);
        if (status != hipSuccess)
        {
            printf("stream_nx_exp destroy failed\n");
            fflush(stdout);
            return 1;
        }
        status = hipStreamDestroy(stream_ny_exp);
        if (status != hipSuccess)
        {
            printf("stream_ny_exp destroy failed\n");
            fflush(stdout);
            return 1;
        }

        hipFree(xyexp_xy);
        if (status != hipSuccess)
        {
            printf("hipFree failed for xyexp_xy\n");
            fflush(stdout);
            return 1;
        }
        hipFree(xyexp_nxy);
        if (status != hipSuccess)
        {
            printf("hipFree failed for xyexp_nxy\n");
            fflush(stdout);
            return 1;
        }
    }

    /* destroy stream */
    status = hipStreamDestroy(stream_y_exp);
    if (status != hipSuccess)
    {
        printf("stream_y_exp destroy failed\n");
        fflush(stdout);
        return 1;
    }

    return 0;
}


#define REGISTER_KERNEL_LAUNCHER(T, V) \
    int instantiateComputeWeightedAverageWirelengthSparseLauncher(\
            const T* x, const T* y, \
            const int* flat_netpin, \
            const int* netpin_start, \
            const T* netpin_values, \
            const int* pin2net_map, \
            const unsigned char* net_mask, \
            int num_nets, \
            int num_pins, \
            const T* inv_gamma, \
            T* exp_xy, T* exp_nxy, \
            T* exp_xy_sum, T* exp_nxy_sum,\
            T* xyexp_xy_sum, T* xyexp_nxy_sum, \
            V* xy_max, V* xy_min, \
            T* partial_wl, \
            const T* grad_tensor, \
            T* grad_x_tensor, T* grad_y_tensor \
            )\
    {\
        return computeWeightedAverageWirelengthCudaSparseLauncher(\
                x, y, \
                flat_netpin, \
                netpin_start, \
                netpin_values, \
                pin2net_map, \
                net_mask, \
                num_nets,\
                num_pins,\
                inv_gamma, \
                exp_xy, exp_nxy, \
                exp_xy_sum, exp_nxy_sum, \
                xyexp_xy_sum, xyexp_nxy_sum, \
                xy_max, xy_min, \
                partial_wl, \
                grad_tensor, \
                grad_x_tensor, grad_y_tensor  \
                );\
    }
REGISTER_KERNEL_LAUNCHER(float, int);
REGISTER_KERNEL_LAUNCHER(double, int);

DREAMPLACE_END_NAMESPACE
