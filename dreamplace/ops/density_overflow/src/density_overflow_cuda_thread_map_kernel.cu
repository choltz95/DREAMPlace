#include "hip/hip_runtime.h"
/**
 * @file   density_overflow_cuda_thread_map_kernel.cu
 * @author Yibo Lin
 * @date   Jun 2018
 * @brief  Compute density map on CUDA with cell2bin parallelization
 */
#include <stdio.h>
#include <math.h>
#include <float.h>
#include "hip/hip_runtime.h"
#include "utility/src/print.h"
#include "utility/src/Msg.h"

DREAMPLACE_BEGIN_NAMESPACE

template <typename T>
__global__ void computeDensityMapWithThreadMap(
        const T* x_tensor, const T* y_tensor,
        const T* node_size_x_tensor, const T* node_size_y_tensor,
        const T* bin_center_x_tensor, const T* bin_center_y_tensor,
        const int* thread2node_map, const int* thread2bin_x_map, const int* thread2bin_y_map,
        const int num_threads,
        const int num_nodes,
        const int num_bins_x, const int num_bins_y,
        const T xl, const T yl, const T xh, const T yh,
        const T bin_size_x, const T bin_size_y,
        T* density_map_tensor)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // rank-one update density map
    if (i < num_threads)
    {
        // density overflow function
        auto computeDensityOverflowFunc = [](T x, T node_size, T bin_center, T bin_size){
            return max(T(0.0), min(x+node_size, bin_center+bin_size/2) - max(x, bin_center-bin_size/2));
        };
        int node_id = thread2node_map[i];
        int bin_offset_x = thread2bin_x_map[i];
        int bin_offset_y = thread2bin_y_map[i];
        // x direction
        int bin_index_xl = int((x_tensor[node_id]-xl)/bin_size_x);
        bin_index_xl = max(bin_index_xl, 0);
        int k = bin_index_xl+bin_offset_x;
        if (k+1 > num_bins_x)
        {
            return;
        }
        // y direction
        int bin_index_yl = int((y_tensor[node_id]-yl)/bin_size_y);
        bin_index_yl = max(bin_index_yl, 0);
        int h = bin_index_yl+bin_offset_y;
        if (h+1 > num_bins_y)
        {
            return;
        }

        T px = computeDensityOverflowFunc(x_tensor[node_id], node_size_x_tensor[node_id], bin_center_x_tensor[k], bin_size_x);
        T py = computeDensityOverflowFunc(y_tensor[node_id], node_size_y_tensor[node_id], bin_center_y_tensor[h], bin_size_y);
        // still area 
        atomicAdd(&density_map_tensor[k*num_bins_y+h], px*py); 
    }
}

template <typename T>
int computeDensityOverflowMapCudaThreadMapLauncher(
        const T* x_tensor, const T* y_tensor,
        const T* node_size_x_tensor, const T* node_size_y_tensor,
        const T* bin_center_x_tensor, const T* bin_center_y_tensor,
        const int* thread2node_map, const int* thread2bin_x_map, const int* thread2bin_y_map,
        const int num_threads,
        const int num_nodes,
        const int num_bins_x, const int num_bins_y,
        const T xl, const T yl, const T xh, const T yh,
        const T bin_size_x, const T bin_size_y,
        T* density_map_tensor
        )
{
    int thread_count = 512;
    int block_count = (num_threads - 1 + thread_count) / thread_count;

    computeDensityMapWithThreadMap<<<block_count, thread_count>>>(
            x_tensor, y_tensor,
            node_size_x_tensor, node_size_y_tensor,
            bin_center_x_tensor, bin_center_y_tensor,
            thread2node_map, thread2bin_x_map, thread2bin_y_map,
            num_threads,
            num_nodes,
            num_bins_x, num_bins_y,
            xl, yl, xh, yh,
            bin_size_x, bin_size_y,
            density_map_tensor);

    return 0;
}

#define REGISTER_KERNEL_LAUNCHER(T) \
    int instantiateComputeDensityOverflowMapLauncher(\
            const T* x_tensor, const T* y_tensor, \
            const T* node_size_x_tensor, const T* node_size_y_tensor, \
            const T* bin_center_x_tensor, const T* bin_center_y_tensor, \
            const int* thread2node_map, const int* thread2bin_x_map, const int* thread2bin_y_map, \
            const int num_threads, \
            const int num_nodes, \
            const int num_bins_x, const int num_bins_y, \
            const T xl, const T yl, const T xh, const T yh, \
            const T bin_size_x, const T bin_size_y, \
            T* density_map_tensor\
            )\
    { \
        return computeDensityOverflowMapCudaThreadMapLauncher(\
                x_tensor, y_tensor, \
                node_size_x_tensor, node_size_y_tensor, \
                bin_center_x_tensor, bin_center_y_tensor, \
                thread2node_map, thread2bin_x_map, thread2bin_y_map, \
                num_threads, \
                num_nodes, \
                num_bins_x, num_bins_y, \
                xl, yl, xh, yh, \
                bin_size_x, bin_size_y, \
                density_map_tensor\
                );\
    }
REGISTER_KERNEL_LAUNCHER(float);
REGISTER_KERNEL_LAUNCHER(double);

DREAMPLACE_END_NAMESPACE
