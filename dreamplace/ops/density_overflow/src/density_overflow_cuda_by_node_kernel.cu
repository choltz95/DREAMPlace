#include "hip/hip_runtime.h"
/**
 * @file   density_overflow_cuda_by_node_kernel.cu
 * @author Yibo Lin
 * @date   Jun 2018
 * @brief  Compute density map on CUDA  
 */
#include <stdio.h>
#include <math.h>
#include <float.h>
#include "hip/hip_runtime.h"
#include "utility/src/print.h"
#include "utility/src/Msg.h"

DREAMPLACE_BEGIN_NAMESPACE

template <typename T>
__global__ void computeDensityMapByNode(
        const T* x_tensor, const T* y_tensor, 
        const T* node_size_x_tensor, const T* node_size_y_tensor, 
        const T* bin_center_x_tensor, const T* bin_center_y_tensor, 
        const int num_nodes, 
        const int num_bins_x, const int num_bins_y, 
        const T xl, const T yl, const T xh, const T yh, 
        const T bin_size_x, const T bin_size_y, 
        T* density_map_tensor) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // rank-one update density map 
    if (i < num_nodes) 
    {
        // density overflow function 
        auto computeDensityOverflowFunc = [](T x, T node_size, T bin_center, T bin_size){
            return max(T(0.0), min(x+node_size, bin_center+bin_size/2) - max(x, bin_center-bin_size/2));
        };
        int node_id = i; 
        // x direction 
        int bin_index_xl = int((x_tensor[node_id]-xl)/bin_size_x);
        bin_index_xl = max(bin_index_xl, 0);
        int bin_index_xh = int((x_tensor[node_id]+node_size_x_tensor[node_id]-xl)/bin_size_x)+1;
        bin_index_xh = min(bin_index_xh, num_bins_x);

        // y direction 
        int bin_index_yl = int((y_tensor[node_id]-yl)/bin_size_y);
        bin_index_yl = max(bin_index_yl, 0);
        int bin_index_yh = int((y_tensor[node_id]+node_size_y_tensor[node_id]-yl)/bin_size_y)+1;
        bin_index_yh = min(bin_index_yh, num_bins_y);

        for (int k = bin_index_xl; k < bin_index_xh; ++k)
        {
            for (int h = bin_index_yl; h < bin_index_yh; ++h)
            {
                T px = computeDensityOverflowFunc(x_tensor[node_id], node_size_x_tensor[node_id], bin_center_x_tensor[k], bin_size_x);
                T py = computeDensityOverflowFunc(y_tensor[node_id], node_size_y_tensor[node_id], bin_center_y_tensor[h], bin_size_y);
                // still area 
                atomicAdd(&density_map_tensor[k*num_bins_y+h], px*py); 
            }
        }
    }
}

template <typename T>
int computeDensityOverflowMapCudaByNodeLauncher(
        const T* x_tensor, const T* y_tensor, 
        const T* node_size_x_tensor, const T* node_size_y_tensor, 
        const T* bin_center_x_tensor, const T* bin_center_y_tensor, 
        const int num_nodes, 
        const int num_bins_x, const int num_bins_y, 
        const T xl, const T yl, const T xh, const T yh, 
        const T bin_size_x, const T bin_size_y, 
        T* density_map_tensor
        )
{
    int thread_count = 512;
    int block_count = (num_nodes - 1 + thread_count) / thread_count;

    computeDensityMapByNode<<<block_count, thread_count>>>(
            x_tensor, y_tensor, 
            node_size_x_tensor, node_size_y_tensor, 
            bin_center_x_tensor, bin_center_y_tensor, 
            num_nodes, 
            num_bins_x, num_bins_y, 
            xl, yl, xh, yh, 
            bin_size_x, bin_size_y, 
            density_map_tensor);

    return 0; 
}

#define REGISTER_KERNEL_LAUNCHER(T) \
    int instantiateComputeDensityOverflowMapByNodeLauncher(\
            const T* x_tensor, const T* y_tensor, \
            const T* node_size_x_tensor, const T* node_size_y_tensor, \
            const T* bin_center_x_tensor, const T* bin_center_y_tensor, \
            const int num_nodes, \
            const int num_bins_x, const int num_bins_y, \
            const T xl, const T yl, const T xh, const T yh, \
            const T bin_size_x, const T bin_size_y, \
            T* density_map_tensor\
            )\
    { \
        return computeDensityOverflowMapCudaByNodeLauncher(\
                x_tensor, y_tensor, \
                node_size_x_tensor, node_size_y_tensor, \
                bin_center_x_tensor, bin_center_y_tensor, \
                num_nodes, \
                num_bins_x, num_bins_y, \
                xl, yl, xh, yh, \
                bin_size_x, bin_size_y, \
                density_map_tensor\
                );\
    }
REGISTER_KERNEL_LAUNCHER(float);
REGISTER_KERNEL_LAUNCHER(double);

DREAMPLACE_END_NAMESPACE
