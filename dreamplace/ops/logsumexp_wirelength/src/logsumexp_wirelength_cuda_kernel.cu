#include "hip/hip_runtime.h"
#include <cfloat>
#include <stdio.h>
#include "assert.h"
#include "hip/hip_runtime.h"
#include "utility/src/csrmv.h"
#include "utility/src/print.h"
#include "utility/src/Msg.h"

DREAMPLACE_BEGIN_NAMESPACE

template <typename T>
__global__ void fillArray(T* x, const int n, const T v)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) 
    {
        x[i] = v; 
    }
}

template <typename T>
__global__ void computeExp(const T* x, const T* nx, const int n, const T* gamma, T* exp_x)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) 
    {
        exp_x[i] = exp(x[i]/(*gamma)); 
    }
}

template <typename T>
__global__ void computeNegExp(const T* x, const T* nx, const int n, const T* gamma, T* exp_nx)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) 
    {
        exp_nx[i] = exp(-x[i]/(*gamma)); 
    }
}

template <typename T>
__global__ void computeMaxAndExp(
        const T* x, 
        const int* flat_netpin, 
        const int* netpin_start, 
        const unsigned char* net_mask, 
        int num_nets,
        const T* gamma, 
        T* x_max, 
        T* exp_x
        )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_nets) 
    {
        x_max[i] = -FLT_MAX; 
        if (net_mask[i])
        {
            for (int j = netpin_start[i]; j < netpin_start[i+1]; ++j)
            {
                int jj = flat_netpin[j];
                T xx = x[jj];
                x_max[i] = max(x_max[i], xx);
            }
            for (int j = netpin_start[i]; j < netpin_start[i+1]; ++j)
            {
                int jj = flat_netpin[j];
                T xx = x[jj];
                exp_x[jj] = exp((xx-x_max[i])/(*gamma)); 
            }
        }
    }
}

template <typename T>
__global__ void computeMinAndNegExp(
        const T* x, 
        const int* flat_netpin, 
        const int* netpin_start, 
        const unsigned char* net_mask, 
        int num_nets,
        const T* gamma, 
        T* x_min, 
        T* exp_nx
        )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_nets) 
    {
        x_min[i] = FLT_MAX; 
        if (net_mask[i])
        {
            for (int j = netpin_start[i]; j < netpin_start[i+1]; ++j)
            {
                int jj = flat_netpin[j];
                T xx = x[jj];
                x_min[i] = min(x_min[i], xx);
            }
            for (int j = netpin_start[i]; j < netpin_start[i+1]; ++j)
            {
                int jj = flat_netpin[j];
                T xx = x[jj];
                exp_nx[jj] = exp(-(xx-x_min[i])/(*gamma)); 
            }
        }
    }
}

template <typename T>
__global__ void computeLogSumExp(
        const T* exp_x_sum, 
        const T* x_max, 
        const int* flat_netpin, 
        const int* netpin_start, 
        const unsigned char* net_mask, 
        int num_nets,
        const T* gamma, 
        T* partial_wl 
        )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_nets) 
    {
        if (net_mask[i])
        {
            partial_wl[i] = (*gamma)*log(exp_x_sum[i]) + x_max[i]; 
        }
        else 
        {
            partial_wl[i] = 0; 
        }
    }
}

template <typename T>
__global__ void computeLogSumNegExp(
        const T* exp_nx_sum, 
        const T* x_min, 
        const int* flat_netpin, 
        const int* netpin_start, 
        const unsigned char* net_mask, 
        int num_nets,
        const T* gamma, 
        T* partial_wl 
        )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_nets) 
    {
        if (net_mask[i])
        {
            partial_wl[i] = (*gamma)*log(exp_nx_sum[i]) - x_min[i]; 
        }
        else 
        {
            partial_wl[i] = 0; 
        }
    }
}

template <typename T>
__global__ void sumArray(const T* x, const int n, T* output)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i == 0)
    {
        *output = 0; 
        for (int j = 0; j < n; ++j)
        {
            *output += x[j];
        }
    }
}

template <typename T>
__global__ void computeLogSumExpWirelengthGrad(
        const T* exp_x, const T* exp_nx, 
        const T* exp_x_sum, const T* exp_nx_sum, 
        const int* flat_netpin, 
        const int* netpin_start, 
        const unsigned char* net_mask, 
        int num_nets,
        const T* gamma, 
        const T* grad_tensor, 
        T* grad_x_tensor
        )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_nets) 
    {
        if (net_mask[i])
        {
            T reciprocal_exp_x_sum = 1.0/exp_x_sum[i]; 
            T reciprocal_exp_nx_sum = 1.0/exp_nx_sum[i]; 
            for (int j = netpin_start[i]; j < netpin_start[i+1]; ++j)
            {
                int jj = flat_netpin[j];
                grad_x_tensor[jj] = (exp_x[jj]*reciprocal_exp_x_sum - exp_nx[jj]*reciprocal_exp_nx_sum)*(*grad_tensor); 
                //grad_x_tensor[jj] = (exp_x[jj]/exp_x_sum[i] - exp_nx[jj]/exp_nx_sum[i])*(*grad_tensor); 
            }
        }
    }
}

template <typename T>
int computeLogSumExpWirelengthCudaLauncher(
        const T* x, const T* y, 
        const int* flat_netpin, 
        const int* netpin_start, 
        const T* netpin_values, 
        const unsigned char* net_mask, 
        int num_nets,
        int num_pins, 
        const T* gamma, 
        T* exp_xy, T* exp_nxy, 
        T* exp_xy_sum, T* exp_nxy_sum, 
        T* partial_wl, // wirelength of each net 
        const T* grad_tensor, 
        T* grad_x_tensor, T* grad_y_tensor // the gradient is partial total wirelength to partial pin position  
        )
{
    int thread_count = 512; 
    int block_count_nets = (num_nets + thread_count - 1) / thread_count; // separate x and y

    hipError_t status; 
    hipStream_t stream_x_exp; 
    hipStream_t stream_nx_exp; 
    hipStream_t stream_y_exp; 
    hipStream_t stream_ny_exp; 
    status = hipStreamCreate(&stream_x_exp);
    if (status != hipSuccess)
    {
        printf("hipStreamCreate failed for stream_x_exp\n");
        fflush(stdout);
        return 1; 
    }
    status = hipStreamCreate(&stream_y_exp);
    if (status != hipSuccess)
    {
        printf("hipStreamCreate failed for stream_y_exp\n");
        fflush(stdout);
        return 1; 
    }

    if (grad_tensor)
    {
        computeLogSumExpWirelengthGrad<<<block_count_nets, thread_count, 0, stream_x_exp>>>(
                exp_xy, exp_nxy, 
                exp_xy_sum, exp_nxy_sum, 
                flat_netpin, 
                netpin_start, 
                net_mask, 
                num_nets,
                gamma, 
                grad_tensor, 
                grad_x_tensor
                );
        computeLogSumExpWirelengthGrad<<<block_count_nets, thread_count, 0, stream_y_exp>>>(
                exp_xy+num_pins, exp_nxy+num_pins, 
                exp_xy_sum+num_nets, exp_nxy_sum+num_nets, 
                flat_netpin, 
                netpin_start, 
                net_mask, 
                num_nets,
                gamma, 
                grad_tensor, 
                grad_y_tensor
                );
    }
    else
    {
        status = hipStreamCreate(&stream_nx_exp);
        if (status != hipSuccess)
        {
            printf("hipStreamCreate failed for stream_nx_exp\n");
            fflush(stdout);
            return 1; 
        }
        status = hipStreamCreate(&stream_ny_exp);
        if (status != hipSuccess)
        {
            printf("hipStreamCreate failed for stream_ny_exp\n");
            fflush(stdout);
            return 1; 
        }

        T* xy_max = nullptr; 
        status = hipMalloc((void**)&xy_max, 2*num_nets*sizeof(T)); 
        if (status != hipSuccess)
        {
            printf("hipMalloc failed for xy_max\n");
            fflush(stdout);
            return 1; 
        }
        T* xy_min = nullptr; 
        status = hipMalloc((void**)&xy_min, 2*num_nets*sizeof(T)); 
        if (status != hipSuccess)
        {
            printf("hipMalloc failed for xy_min\n");
            fflush(stdout);
            return 1; 
        }

        //T* partial_wl = nullptr; 
        //status = hipMalloc((void**)&partial_wl, 2*num_nets*sizeof(T)); 
        //if (status != hipSuccess)
        //{
        //    printf("hipMalloc failed for partial_wl\n");
        //    fflush(stdout);
        //    return 1; 
        //}
        //// be careful, partial_wl is not initialized yet 

        T alpha = 1.0; 
        T beta = 0.0; 

        computeMaxAndExp<<<block_count_nets, thread_count, 0, stream_x_exp>>>(
                x, 
                flat_netpin, 
                netpin_start, 
                net_mask, 
                num_nets, 
                gamma, 
                xy_max,
                exp_xy
                );
        computeMinAndNegExp<<<block_count_nets, thread_count, 0, stream_nx_exp>>>(
                x, 
                flat_netpin, 
                netpin_start, 
                net_mask, 
                num_nets, 
                gamma, 
                xy_min, 
                exp_nxy
                );
        computeMaxAndExp<<<block_count_nets, thread_count, 0, stream_y_exp>>>(
                y, 
                flat_netpin, 
                netpin_start, 
                net_mask, 
                num_nets, 
                gamma, 
                xy_max+num_nets,
                exp_xy+num_pins
                );
        computeMinAndNegExp<<<block_count_nets, thread_count, 0, stream_ny_exp>>>(
                y, 
                flat_netpin, 
                netpin_start, 
                net_mask, 
                num_nets, 
                gamma, 
                xy_min+num_nets, 
                exp_nxy+num_pins
                );

        hipsparseStatus_t sparse_status;
        hipsparseHandle_t handle_x_exp = 0;
        hipsparseHandle_t handle_nx_exp = 0;
        hipsparseHandle_t handle_y_exp = 0;
        hipsparseHandle_t handle_ny_exp = 0;
        hipsparseMatDescr_t descr = 0;
        /* initialize cusparse library */
        sparse_status= hipsparseCreate(&handle_x_exp);
        if (sparse_status != HIPSPARSE_STATUS_SUCCESS) {
            printf("CUSPARSE Library initialization failed\n");
            fflush(stdout);
            return 1;
        }
        sparse_status= hipsparseCreate(&handle_nx_exp);
        if (sparse_status != HIPSPARSE_STATUS_SUCCESS) {
            printf("CUSPARSE Library initialization failed\n");
            fflush(stdout);
            return 1;
        }
        sparse_status= hipsparseCreate(&handle_y_exp);
        if (sparse_status != HIPSPARSE_STATUS_SUCCESS) {
            printf("CUSPARSE Library initialization failed\n");
            fflush(stdout);
            return 1;
        }
        sparse_status= hipsparseCreate(&handle_ny_exp);
        if (sparse_status != HIPSPARSE_STATUS_SUCCESS) {
            printf("CUSPARSE Library initialization failed\n");
            fflush(stdout);
            return 1;
        }
        /* create and setup matrix descriptor */ 
        sparse_status= hipsparseCreateMatDescr(&descr); 
        if (sparse_status != HIPSPARSE_STATUS_SUCCESS) {
            printf("Matrix descriptor initialization failed\n");
            fflush(stdout);
            return 1;
        } 
        hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);  
        hipsparseSetStream(handle_x_exp, stream_x_exp);
        hipsparseSetStream(handle_nx_exp, stream_nx_exp);
        hipsparseSetStream(handle_y_exp, stream_y_exp);
        hipsparseSetStream(handle_ny_exp, stream_ny_exp);

        csrmv(
                handle_x_exp, 
                HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                num_nets, 
                num_pins, 
                num_pins, 
                &alpha, 
                descr, 
                netpin_values, 
                netpin_start, flat_netpin, 
                exp_xy, 
                &beta, 
                exp_xy_sum
                ); 
        csrmv(
                handle_y_exp, 
                HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                num_nets, 
                num_pins, 
                num_pins, 
                &alpha, 
                descr, 
                netpin_values, 
                netpin_start, flat_netpin, 
                exp_xy+num_pins, 
                &beta, 
                exp_xy_sum+num_nets
                ); 
        csrmv(
                handle_nx_exp, 
                HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                num_nets, 
                num_pins, 
                num_pins, 
                &alpha, 
                descr, 
                netpin_values, 
                netpin_start, flat_netpin, 
                exp_nxy, 
                &beta, 
                exp_nxy_sum
                ); 
        csrmv(
                handle_ny_exp, 
                HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                num_nets, 
                num_pins, 
                num_pins, 
                &alpha, 
                descr, 
                netpin_values, 
                netpin_start, flat_netpin, 
                exp_nxy+num_pins, 
                &beta, 
                exp_nxy_sum+num_nets
                ); 

        computeLogSumExp<<<block_count_nets, thread_count, 0, stream_x_exp>>>(
                exp_xy_sum, 
                xy_max, 
                flat_netpin, 
                netpin_start, 
                net_mask, 
                num_nets,
                gamma, 
                partial_wl
                );
        computeLogSumNegExp<<<block_count_nets, thread_count, 0, stream_nx_exp>>>(
                exp_nxy_sum, 
                xy_min, 
                flat_netpin, 
                netpin_start, 
                net_mask, 
                num_nets,
                gamma, 
                partial_wl+num_nets
                );

        computeLogSumExp<<<block_count_nets, thread_count, 0, stream_y_exp>>>(
                exp_xy_sum+num_nets, 
                xy_max+num_nets, 
                flat_netpin, 
                netpin_start, 
                net_mask, 
                num_nets,
                gamma, 
                partial_wl+2*num_nets
                );
        computeLogSumNegExp<<<block_count_nets, thread_count, 0, stream_ny_exp>>>(
                exp_nxy_sum+num_nets, 
                xy_min+num_nets, 
                flat_netpin, 
                netpin_start, 
                net_mask, 
                num_nets,
                gamma, 
                partial_wl+3*num_nets
                );

        /* destroy matrix descriptor */ 
        sparse_status = hipsparseDestroyMatDescr(descr); 
        descr = 0;
        if (sparse_status != HIPSPARSE_STATUS_SUCCESS) 
        {
            printf("Matrix descriptor destruction failed\n");
            fflush(stdout);
            return 1;
        }

        /* destroy handle */
        sparse_status = hipsparseDestroy(handle_x_exp);
        handle_x_exp = 0;
        if (sparse_status != HIPSPARSE_STATUS_SUCCESS) 
        {
            printf("CUSPARSE Library release of resources failed\n");
            fflush(stdout);
            return 1;
        }   
        sparse_status = hipsparseDestroy(handle_nx_exp);
        handle_nx_exp = 0;
        if (sparse_status != HIPSPARSE_STATUS_SUCCESS) 
        {
            printf("CUSPARSE Library release of resources failed\n");
            fflush(stdout);
            return 1;
        }   
        sparse_status = hipsparseDestroy(handle_y_exp);
        handle_y_exp = 0;
        if (sparse_status != HIPSPARSE_STATUS_SUCCESS) 
        {
            printf("CUSPARSE Library release of resources failed\n");
            fflush(stdout);
            return 1;
        }   
        sparse_status = hipsparseDestroy(handle_ny_exp);
        handle_ny_exp = 0;
        if (sparse_status != HIPSPARSE_STATUS_SUCCESS) 
        {
            printf("CUSPARSE Library release of resources failed\n");
            fflush(stdout);
            return 1;
        }   

        // I move out the summation to use ATen 
        // significant speedup is observed 
        //sumArray<<<1, 1>>>(partial_wl, 2*num_nets, wl);

        if (xy_max)
        {
            hipFree(xy_max); 
            xy_max = nullptr; 
        }
        if (xy_min)
        {
            hipFree(xy_min); 
            xy_min = nullptr; 
        }
        //if (partial_wl)
        //{
        //    hipFree(partial_wl);
        //    partial_wl = nullptr; 
        //}
        fflush(stdout);

        status = hipStreamDestroy(stream_nx_exp); 
        if (status != hipSuccess) 
        {
            printf("stream_nx_exp destroy failed\n");
            fflush(stdout);
            return 1;
        }   
        status = hipStreamDestroy(stream_ny_exp);  
        if (status != hipSuccess) 
        {
            printf("stream_ny_exp destroy failed\n");
            fflush(stdout);
            return 1;
        }   
    }

    /* destroy stream */
    status = hipStreamDestroy(stream_x_exp); 
    if (status != hipSuccess) 
    {
        printf("stream_x_exp destroy failed\n");
        fflush(stdout);
        return 1;
    }   
    status = hipStreamDestroy(stream_y_exp); 
    if (status != hipSuccess) 
    {
        printf("stream_y_exp destroy failed\n");
        fflush(stdout);
        return 1;
    }   

    return 0; 
}


#define REGISTER_KERNEL_LAUNCHER(T) \
    int instantiateComputeLogSumExpWirelengthLauncher(\
            const T* x, const T* y, \
            const int* flat_netpin, \
            const int* netpin_start, \
            const T* netpin_values, \
            const unsigned char* net_mask, \
            int num_nets,\
            int num_pins,\
            const T* gamma, \
            T* exp_xy, T* exp_nxy, \
            T* exp_xy_sum, T* exp_nxy_sum, \
            T* partial_wl, \
            const T* grad_tensor, \
            T* grad_x_tensor, T* grad_y_tensor  \
            )\
    {\
        return computeLogSumExpWirelengthCudaLauncher(\
                x, y, \
                flat_netpin, \
                netpin_start, \
                netpin_values, \
                net_mask, \
                num_nets,\
                num_pins,\
                gamma, \
                exp_xy, exp_nxy, \
                exp_xy_sum, exp_nxy_sum, \
                partial_wl, \
                grad_tensor, \
                grad_x_tensor, grad_y_tensor  \
                );\
    }
REGISTER_KERNEL_LAUNCHER(float);
REGISTER_KERNEL_LAUNCHER(double);

DREAMPLACE_END_NAMESPACE
